#include <flashinfer/gemm/gemm_fuse_broadcast.cuh>

#include "pytorch_extension_utils.h"

using namespace flashinfer;
using namespace flashinfer::gemm_fuse_broadcast;

void CutlassGemmFuseBroadcast(at::Tensor A, at::Tensor B, std::vector<at::Tensor> C, int rank,
    int64_t hip_stream) {

hipStream_t stream = reinterpret_cast<hipStream_t>(hip_stream);
DISPATCH_PYTORCH_DTYPE_TO_CTYPE(A.scalar_type(), c_type, [&] {
    using cutlass_t = typename cutlass_dtype<c_type>::value;
    std::vector<void *> c_ptr;
    for(const auto& t : C){
        c_ptr.push_back(t.data_ptr());
    }
    auto status = CutlassGemmFuseBroadcastRun<cutlass_t>(
        A.data_ptr(), B.data_ptr(), c_ptr, rank, stream);
    TORCH_CHECK(status == hipSuccess,
                "Failed to run CutlassGemmFuseBroadcast: ", hipGetErrorString(status));
    return true;
});
}
 